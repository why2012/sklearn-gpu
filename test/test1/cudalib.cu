#include "hip/hip_runtime.h"
#include "cudalib.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#ifdef _WIN32
	#include <windows.h>
#else
	#include <sys/time.h>
#endif
#ifdef WIN32
	int gettimeofday(struct timeval* tp, void* tzp) {
		time_t clock;
		struct tm tm;
		SYSTEMTIME wtm;
		GetLocalTime(&wtm);
		tm.tm_year = wtm.wYear - 1900;
		tm.tm_mon = wtm.wMonth - 1;
		tm.tm_mday = wtm.wDay;
		tm.tm_hour = wtm.wHour;
		tm.tm_min = wtm.wMinute;
		tm.tm_sec = wtm.wSecond;
		tm.tm_isdst = -1;
		clock = mktime(&tm);
		tp->tv_sec = clock;
		tp->tv_usec = wtm.wMilliseconds * 1000;
		return 0;
	}
#endif

#define CHECK(call)															\
{																			\
	const hipError_t error = call;											\
	if (error != hipSuccess) {												\
		printf("Error: %s:%d, ", __FILE__, __LINE__);						\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));	\
		exit(EXIT_FAILURE);													\
	}																		\
}

double cpuSeconds() {
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

__global__ void _checkIndex() {
	printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
		threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
		gridDim.x, gridDim.y, gridDim.z);
}

void checkIndex() {
	printf("GPU Side Invocation Check.\n");
	dim3 block(3);
	dim3 grid((6 + block.x - 1) / block.x);
	double iStart = cpuSeconds();
	_checkIndex<<<grid, block>>>();
	hipDeviceSynchronize();
	double iEnd = cpuSeconds();
	printf("checkIndex<<<grid, block>>> time elapsed %f.", iEnd - iStart);
	hipDeviceReset();
}

void printDeviceProp() {
	int deviceCount = 0;
	CHECK(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		printf("There are no available devices that support CUDA\n");
	} else {
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}
	int dev = 0, driverVersion = 0, runtimeVersion = 0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("Device: %d, \"%s\"\n", dev, deviceProp.name);
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf("	CUDA Driver Version / RuntimeVersion 	%d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, 
		runtimeVersion / 1000, (runtimeVersion % 100) / 10);
	printf("	Total amount of global memory:	%.2f GB (%llu bytes)\n", (float)deviceProp.totalGlobalMem / (pow(1024.0, 3)),
		(unsigned long long) deviceProp.totalGlobalMem);
	printf("	Multiprocessor Count: %d\n", deviceProp.multiProcessorCount);
	printf("	GPU Clock rate:	%.0f MHz(%.2f GHz)\n", deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
	printf("	Memory Clock rate:	%.0f MHz\n", deviceProp.memoryClockRate * 1e-3f);
	printf("	Memory Bus Width:	%d-bit\n", deviceProp.memoryBusWidth);
	printf("	Warp size:	%d\n", deviceProp.warpSize);
	printf("	Maximum number of threads per multiprocessor: %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("	Maximum number of threads per block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("	Maximum size of each dimension of a block: %d x %d x %d\n", deviceProp.maxThreadsDim[0], 
		deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("	Maximum size of each dimension of a grid: %d x %d x %d\n", deviceProp.maxGridSize[0], 
		deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
}

__global__ void reduceNeighbor(int* g_idata, int* g_odata, unsigned int n) {
	unsigned int tid = threadIdx.x;
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int* idata = g_idata + blockIdx.x * blockDim.x;
	if (idx >= n) return;
	for (int stride = 1; stride < blockDim.x; stride *= 2) {
		if ((tid % (2 * stride)) == 0) {
			idata[tid] += idata[tid + stride];
		}
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

__global__ void reduceNeighborPlus(int* g_idata, int* g_odata, unsigned int n) {
	unsigned int tid = threadIdx.x;
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int* idata = g_idata + blockIdx.x * blockDim.x;
	if (idx >= n) return;
	for (int stride = 1; stride < blockDim.x; stride *= 2) {
		int index = 2 * stride * tid;
		if (index < blockDim.x) {
			idata[index] += idata[index + stride];
		}
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

void __reduceCheck(int blockSize = 512) {
	int size = 1 << 24;
	dim3 block(blockSize);
	dim3 grid((size + block.x - 1) / block.x);
	int gpu_sum = 0;
	size_t bytes = size * sizeof(int);
	int* h_idata = (int*)malloc(bytes);
	int* h_odata = (int*)malloc(grid.x * sizeof(int));
	for (int i = 0; i < size; i++) {
		h_idata[i] = (int)(rand() & 0xff);
	}
	int* d_idata = NULL;
	int* d_odata = NULL;
	hipMalloc((void**)&d_idata, bytes);
	hipMalloc((void**)&d_odata, grid.x * sizeof(int));

	hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
	double iStart = cpuSeconds();
	reduceNeighbor<<<grid, block>>>(d_idata, d_odata, size);
	hipDeviceSynchronize();
	double iEnd = cpuSeconds();
	hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
	printf("gpu Neighbored Reduce elapsed %.2f s gpu_sum: %d <<<grid %d block %d>>>\n", iEnd - iStart, gpu_sum, grid.x, block.x);
	
	hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
	iStart = cpuSeconds();
	reduceNeighborPlus<<<grid, block>>>(d_idata, d_odata, size);
	hipDeviceSynchronize();
	iEnd = cpuSeconds();
	hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
	gpu_sum = 0;
	for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];
	printf("gpu Neighbored Reduce Plus elapsed %.2f s gpu_sum: %d <<<grid %d block %d>>>\n", iEnd - iStart, gpu_sum, grid.x, block.x);
	
	free(h_idata);
	free(h_odata);
	hipFree(d_idata);
	hipFree(d_odata);
	hipDeviceReset();
}

int cuCheck() {
	printDeviceProp();
	printf("Reduce Check\n");
	__reduceCheck();
	return 0;
}