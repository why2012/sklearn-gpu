#include "hip/hip_runtime.h"
#include "cudalib.h"
#include <stdio.h>

float a_triple(float a) {
	return a * a * a;
}

float a_triple_plus_10(float a) {
	return a * a * a + 10;
}

__global__ void helloworld_gpu() {
	printf("Hello World from GPU!\n");
}

void helloworld() {
	printf("Hello World from CPU!\n");
	helloworld_gpu<<<1,10>>>();
	hipDeviceReset();
}